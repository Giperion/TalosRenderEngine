#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include "math_functions.h"
#include "EngineCuda_C.cuh"

extern "C"
{
	//Как тут в вашем С устроить объекты, а?...
	//global for all
	typedef unsigned int DeviceToken;
	typedef int EngineStatus;

	struct EngineData
	{
		DeviceToken ActiveDevice;

		int DeviceCount;
		int ComputeDeviceCount;
		DeviceToken* Devices;

		//minimal specs
		unsigned int ComputeMajor;
		unsigned int ComputeMinor;
	};

	EngineData DataDesc;

	EngineStatus status;
	hipError_t errcode;
	wchar_t* DebugMessage;

	int BestDeviceID;

	void* argsMemTest;


	int tempDevices[1024];

	/*
	* По текущей реализации ширина кадра формируется значением максимального числа потоков на блок
	* Все потому что пока что задача кидается по типу: Одна линия кадра = Один блок
	* Что позволяет по максимуму создать блоки и потоки за один вызов.
	*/

	__device__ int getGlobalIdx_1D_2D()
	{
			return blockIdx.x * blockDim.x * blockDim.y
				+ threadIdx.y * blockDim.x + threadIdx.x;
	}

	__device__ int getGlobalIdx_2D_2D()
	{
			int blockId = blockIdx.x
				+ blockIdx.y * gridDim.x;
			int threadId = blockId * (blockDim.x * blockDim.y)
				+ (threadIdx.y * blockDim.x)
				+ threadIdx.x;
			return threadId;
		}

	static inline void FastZeroMemory(void* dst, size_t Size)
	{
		int Method = 0;

		//memory aligned?
		if (!(Size % 8))
		{
			Method = 1;
		}

		if (!(Size % 4) && Method == 0)
		{
			Method = 2;
		}

		int IterCount = Size;
		byte* byteMem = (byte*)dst;
		__int64* qwordMem = (__int64*)dst;
		__int32* dwordMem = (__int32*)dst;

		switch (Method)
		{
		case 0:
			for (int i = 0; i < IterCount; i++)
			{
				*byteMem = 0;
				byteMem++;
			}
			break;
		case 1:
			IterCount /= 8;
			for (int i = 0; i < IterCount; i++)
			{
				*qwordMem = 0;
				qwordMem++;
			}
			break;
		case 2:
			IterCount /= 4;
			for (int i = 0; i < IterCount; i++)
			{
				*dwordMem = 0;
				dwordMem++;
			}
			break;
		default:
			break;
		}
	}
	
	const char* GetErrorString(hipError_t errcode)
	{
		return hipGetErrorString(errcode);
	}


	bool cuda_init()
	{
		status = CENGINE_STATUS_INIT;
		DebugMessage = L" ";
		argsMemTest = nullptr;
		//Determine if we have CUDA device?
		hipDeviceProp_t deviceProp;
		errcode = hipGetDeviceCount(&DataDesc.DeviceCount);
		if (errcode == hipErrorInsufficientDriver || errcode == hipErrorNoDevice)
		{
			if (errcode == hipErrorInsufficientDriver)
			{
				DebugMessage = L"Cuda driver is outdated! Please update you graphic card driver and try again!";
			}
			else
			{
				DebugMessage = L"There is no cuda device in this machine!";
			}

			status = CENGINE_STATUS_NOCUDA;
			return false;
		}
		//set compute model to max. Later we update that when looking devices
		DataDesc.ComputeMajor = 0x0000FFFF;
		DataDesc.ComputeMinor = 0x000000FF;

		//errcode = hipHostMalloc(&DataDesc.Devices, sizeof(DeviceToken)*DataDesc.DeviceCount);
		DataDesc.Devices = (DeviceToken*)&tempDevices[0];
		for (int DeviceID = 0; DeviceID < DataDesc.DeviceCount; DeviceID++)
		{
			errcode = hipGetDeviceProperties(&deviceProp, DeviceID);
			if (deviceProp.computeMode != hipComputeMode::hipComputeModeProhibited)
			{
				DataDesc.ComputeDeviceCount++;
			}
			else continue;

			int ComputeVer = (deviceProp.major * 10) + deviceProp.minor;
			int KnownComputeVer = (DataDesc.ComputeMajor * 10) + DataDesc.ComputeMinor;
			if (ComputeVer < KnownComputeVer)
			{
				DataDesc.ComputeMajor = deviceProp.major;
				DataDesc.ComputeMinor = deviceProp.minor;
			}

			DataDesc.Devices[DataDesc.ComputeDeviceCount - 1] = DeviceID;
		}
		if (DataDesc.ComputeDeviceCount == 0)
		{
			if (deviceProp.computeMode == hipComputeMode::hipComputeModeProhibited)
			{
				status = CENGINE_STATUS_FATALERROR;
				DebugMessage = L"Device computeMode set to prohibiten! Can't compute!";
				return false;
			}
		}
		errcode = hipSetDevice(DataDesc.Devices[0]);
		//Get device stuff
		status = CENGINE_STATUS_IDLE;
		return true;
	}



	cudaImage cuda_AllocTexture(int width, int height)
	{
		cudaImage result;
		result.width = width;
		result.height = height;
		result.bytes = (width * height) * 3;
		
		status = CENGINE_STATUS_IDLE;
		return result;
	}


	void cuda_deinit()
	{
		if (status != CENGINE_STATUS_IDLE)
		{
			//TODO Sync code
		}
	}

	void cuda_execCode(char* code)
	{

	}
	void WaitCudaThread()
	{
		hipDeviceSynchronize();
	}

	hipError_t temp_callKernels(int width, int height, pFrame frame, void* args, int argsSize)
	{
		///#TODO: Different devices can have different recommeded grid
		dim3 blocks(4, 4, 1);
		dim3 grid(width / blocks.x, height / blocks.y, 1);

		//dim3 grid(256, 256, 1);
		//dim3 blocks(width / grid.x, height / grid.y, 1);

		errcode = hipConfigureCall(grid, blocks);

		errcode = hipSetupArgument(&width, sizeof(int), 0);
		errcode = hipSetupArgument(&height, sizeof(int), sizeof(int));
		errcode = hipSetupArgument(&frame, sizeof(pFrame), sizeof(int) * 2);
		if (args != nullptr)
		{
			if (argsMemTest == nullptr)
			{
				errcode = hipMalloc(&argsMemTest, argsSize);
			}
			errcode = hipMemcpy(argsMemTest, args, argsSize, hipMemcpyKind::hipMemcpyHostToDevice);
			errcode = hipSetupArgument(&argsMemTest, sizeof(void*), (sizeof(int) * 2) + sizeof(pFrame));
		}


		errcode = hipLaunchByPtr(testKernelFunc);
		return errcode;
	}

	__device__ inline int Lerp(int start, int end, double value)
	{
		return start + (end - start) * value;
	}

	//dim3 blockIdx <- gridDim
	//dim3 threadIdx <- blocks

	__device__ FColor HSV2RGB(double Hue, double Saturation, double Value)
	{
		double      hh, p, q, t, ff;
		long        i;
		FColor        out;
		out.A = 1.0;

		if (Saturation <= 0.0) {       // < is bogus, just shuts up warnings
			out.R = Value;
			out.G = Value;
			out.B = Value;
			return out;
		}
		hh = Hue;
		if (hh >= 360.0) hh = 0.0;
		hh /= 60.0;
		i = (long)hh;
		ff = hh - i;
		p = Value * (1.0 - Saturation);
		q = Value * (1.0 - (Saturation * ff));
		t = Value * (1.0 - (Saturation * (1.0 - ff)));

		switch (i) {
		case 0:
			out.R = Value;
			out.G = t;
			out.B = p;
			break;
		case 1:
			out.R = q;
			out.G = Value;
			out.B = p;
			break;
		case 2:
			out.R = p;
			out.G = Value;
			out.B = t;
			break;

		case 3:
			out.R = p;
			out.G = q;
			out.B = Value;
			break;
		case 4:
			out.R = t;
			out.G = p;
			out.B = Value;
			break;
		case 5:
		default:
			out.R = Value;
			out.G = p;
			out.B = q;
			break;
		}
		return out;
	}

	__device__ Color FColor2Color(FColor inColor)
	{
		Color Out;

		Out.R = Lerp(0, 255, inColor.R);
		Out.G = Lerp(0, 255, inColor.G);
		Out.B = Lerp(0, 255, inColor.B);
		Out.A = Lerp(0, 255, inColor.A);

		return Out;
	}

	__global__ void testKernelFunc(int width, int height, pFrame frame, void* args)
	{
		//init pointers and pixel coord
		//compute target pixel index for thread and block
		int posX = threadIdx.x + (blockIdx.x * blockDim.x);
		int posY = threadIdx.y + (blockIdx.y * blockDim.y);
		//posY += StartY;

		Color* mainFrame = (Color*)frame;
		MandelbrotView* mView = (MandelbrotView*)args;
		double Scale = mView->scale;

		int newIndex = (posY * width) + posX;

		Color& target = mainFrame[newIndex];


		int x = posX;
		int y = posY;

		double centerX = -0.5;
		double centerY = 0.5;

		double ResultX;
		double ResultY;

		double Zx = 0;
		double Zy = 0;
		double Zx_x2 = 0;
		double Zy_x2 = 0;

		const double MinimumResultX = centerX - Scale;
		const double MaximumResultX = centerX + Scale;
		const double MinimumResultY = centerY - Scale;
		const double MaximumResultY = centerY + Scale;

		
		double PixelWidth = (MaximumResultX - MinimumResultX) / width;
		double PixelHeight = (MaximumResultY - MinimumResultY) / height;


		int Iteration = 0;
		int MaxIteration = mView->iteration;
		
		const double EscapeRadius = 2.0L;
		double EscapeRadius_x2 = EscapeRadius * EscapeRadius;

		ResultX = (MinimumResultX + PixelWidth * x) + mView->x;
		ResultY = (MinimumResultY + PixelHeight * y) + mView->y;

		if (fabs(ResultY) < PixelHeight / 2) ResultY = 0.0;

        for (; Iteration < MaxIteration && ((Zx_x2 + Zy_x2) < EscapeRadius_x2); Iteration++)
        {
            Zy = 2 * Zx * Zy + ResultY;
            Zx = Zx_x2 - Zy_x2 + ResultX;
            Zx_x2 = Zx * Zx;
            Zy_x2 = Zy * Zy;
        }

		double NewValue = 1.0 * (Iteration % mView->iteration);
		FColor HDRColor = HSV2RGB(Iteration % 361, 0.01, NewValue);
		target = FColor2Color(HDRColor);
	}

	__global__ void JuliaKernelFunc(int width, int height, pFrame frame, void* args)
	{
		int posX = threadIdx.x + (blockIdx.x * blockDim.x);
		int posY = threadIdx.y + (blockIdx.y * blockDim.y);
		//posY += StartY;

		Color* mainFrame = (Color*)frame;
		MandelbrotView* mView = (MandelbrotView*)args;

		double cRe, cIm;
		double newRe, newIm, oldRe, oldIm;
		int newIndex = (posY * width) + posX;

		Color& target = mainFrame[newIndex];

		//pick some values for the constant c, this determines the shape of the Julia Set
		cRe = -0.7;
		cIm = 0.27015;

		double ScaledX = mView->x * 0.03;
		ScaledX /= mView->scale * 2;
		double ScaledY = mView->y * 0.03;
		ScaledY /= mView->scale * 2;

		//calculate the initial real and imaginary part of z, based on the pixel location and zoom and position values
		newRe = 1.5 * (posX - width / 2) / (0.5 * mView->scale * width) + ScaledX;
		newIm = (posY - height / 2) / (0.5 * mView->scale * height) + ScaledY;

		//i will represent the number of iterations
		int i;
		//start the iteration process
		for (i = 0; i < mView->iteration; i++)
		{
			//remember value of previous iteration
			oldRe = newRe;
			oldIm = newIm;
			//the actual iteration, the real and imaginary part are calculated
			newRe = oldRe * oldRe - oldIm * oldIm + cRe;
			newIm = 2 * oldRe * oldIm + cIm;
			//if the point is outside the circle with radius 2: stop
			if ((newRe * newRe + newIm * newIm) > 4) break;
		}

		FColor FResult = HSV2RGB((double)(i % 361), 0.8, 1.0 * (i < mView->iteration));

		target = FColor2Color(FResult);
	}

}