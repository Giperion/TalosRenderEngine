#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include "math_functions.h"
#include "EngineCuda_C.cuh"

extern "C"
{
	//Как тут в вашем С устроить объекты, а?...
	//global for all
	typedef unsigned int DeviceToken;
	typedef int EngineStatus;

	struct EngineData
	{
		DeviceToken ActiveDevice;

		int DeviceCount;
		int ComputeDeviceCount;
		DeviceToken* Devices;

		//minimal specs
		unsigned int ComputeMajor;
		unsigned int ComputeMinor;
	};

	EngineData DataDesc;

	EngineStatus status;
	hipError_t errcode;
	wchar_t* DebugMessage;

	int BestDeviceID;

	void* argsMemTest;


	int tempDevices[1024];

	/*
	* По текущей реализации ширина кадра формируется значением максимального числа потоков на блок
	* Все потому что пока что задача кидается по типу: Одна линия кадра = Один блок
	* Что позволяет по максимуму создать блоки и потоки за один вызов.
	*/

	__device__ int getGlobalIdx_1D_2D()
	{
			return blockIdx.x * blockDim.x * blockDim.y
				+ threadIdx.y * blockDim.x + threadIdx.x;
	}

	__device__ int getGlobalIdx_2D_2D()
	{
			int blockId = blockIdx.x
				+ blockIdx.y * gridDim.x;
			int threadId = blockId * (blockDim.x * blockDim.y)
				+ (threadIdx.y * blockDim.x)
				+ threadIdx.x;
			return threadId;
		}

	static inline void FastZeroMemory(void* dst, size_t Size)
	{
		int Method = 0;

		//memory aligned?
		if (!(Size % 8))
		{
			Method = 1;
		}

		if (!(Size % 4) && Method == 0)
		{
			Method = 2;
		}

		int IterCount = Size;
		byte* byteMem = (byte*)dst;
		__int64* qwordMem = (__int64*)dst;
		__int32* dwordMem = (__int32*)dst;

		switch (Method)
		{
		case 0:
			for (int i = 0; i < IterCount; i++)
			{
				*byteMem = 0;
				byteMem++;
			}
			break;
		case 1:
			IterCount /= 8;
			for (int i = 0; i < IterCount; i++)
			{
				*qwordMem = 0;
				qwordMem++;
			}
			break;
		case 2:
			IterCount /= 4;
			for (int i = 0; i < IterCount; i++)
			{
				*dwordMem = 0;
				dwordMem++;
			}
			break;
		default:
			break;
		}
	}
	
	const char* GetErrorString(hipError_t errcode)
	{
		return hipGetErrorString(errcode);
	}


	bool cuda_init()
	{
		status = CENGINE_STATUS_INIT;
		DebugMessage = L" ";
		argsMemTest = nullptr;
		//Determine if we have CUDA device?
		hipDeviceProp_t deviceProp;
		errcode = hipGetDeviceCount(&DataDesc.DeviceCount);
		if (errcode == hipErrorInsufficientDriver || errcode == hipErrorNoDevice)
		{
			if (errcode == hipErrorInsufficientDriver)
			{
				DebugMessage = L"Cuda driver is outdated! Please update you graphic card driver and try again!";
			}
			else
			{
				DebugMessage = L"There is no cuda device in this machine!";
			}

			status = CENGINE_STATUS_NOCUDA;
			return false;
		}
		//set compute model to max. Later we update that when looking devices
		DataDesc.ComputeMajor = 0x0000FFFF;
		DataDesc.ComputeMinor = 0x000000FF;

		//errcode = hipHostMalloc(&DataDesc.Devices, sizeof(DeviceToken)*DataDesc.DeviceCount);
		DataDesc.Devices = (DeviceToken*)&tempDevices[0];
		for (int DeviceID = 0; DeviceID < DataDesc.DeviceCount; DeviceID++)
		{
			errcode = hipGetDeviceProperties(&deviceProp, DeviceID);
			if (deviceProp.computeMode != hipComputeMode::hipComputeModeProhibited)
			{
				DataDesc.ComputeDeviceCount++;
			}
			else continue;

			int ComputeVer = (deviceProp.major * 10) + deviceProp.minor;
			int KnownComputeVer = (DataDesc.ComputeMajor * 10) + DataDesc.ComputeMinor;
			if (ComputeVer < KnownComputeVer)
			{
				DataDesc.ComputeMajor = deviceProp.major;
				DataDesc.ComputeMinor = deviceProp.minor;
			}

			DataDesc.Devices[DataDesc.ComputeDeviceCount - 1] = DeviceID;
		}
		if (DataDesc.ComputeDeviceCount == 0)
		{
			if (deviceProp.computeMode == hipComputeMode::hipComputeModeProhibited)
			{
				status = CENGINE_STATUS_FATALERROR;
				DebugMessage = L"Device computeMode set to prohibiten! Can't compute!";
				return false;
			}
		}
		errcode = hipSetDevice(DataDesc.Devices[0]);
		//Get device stuff
		status = CENGINE_STATUS_IDLE;
		return true;
	}



	cudaImage cuda_AllocTexture(int width, int height)
	{
		cudaImage result;
		result.width = width;
		result.height = height;
		result.bytes = (width * height) * 3;
		
		status = CENGINE_STATUS_IDLE;
		return result;
	}


	void cuda_deinit()
	{
		if (status != CENGINE_STATUS_IDLE)
		{
			//TODO Sync code
		}
	}

	void cuda_execCode(char* code)
	{

	}
	void WaitCudaThread()
	{
		hipDeviceSynchronize();
	}

	hipError_t temp_callKernels(int width, int height, pFrame frame, void* args, int argsSize)
	{
		///#TODO: Different devices can have different recommeded grid
		//dim3 blocks(4, 4, 1);
		//dim3 grid(BlockWidth / blocks.x, BlockHeight / blocks.y, 1);

		dim3 grid(256, 256, 1);
		dim3 blocks(width / grid.x, height / grid.y, 1);

		errcode = hipConfigureCall(grid, blocks);

		errcode = hipSetupArgument(&width, sizeof(int), 0);
		errcode = hipSetupArgument(&height, sizeof(int), sizeof(int));
		errcode = hipSetupArgument(&frame, sizeof(pFrame), sizeof(int) * 2);
		if (args != nullptr)
		{
			if (argsMemTest == nullptr)
			{
				errcode = hipMalloc(&argsMemTest, argsSize);
			}
			errcode = hipMemcpy(argsMemTest, args, argsSize, hipMemcpyKind::hipMemcpyHostToDevice);
			errcode = hipSetupArgument(&argsMemTest, sizeof(void*), (sizeof(int) * 2) + sizeof(pFrame));
		}


		errcode = hipLaunchByPtr(testKernelFunc);
		return errcode;
	}

	__device__ inline int Lerp(int start, int end, double value)
	{
		return start + (end - start) * value;
	}

	//dim3 blockIdx <- gridDim
	//dim3 threadIdx <- blocks

	__global__ void testKernelFunc(int width, int height, pFrame frame, void* args)
	{
		//init pointers and pixel coord
		//compute target pixel index for thread and block
		int posX = threadIdx.x + (blockIdx.x * blockDim.x);
		int posY = threadIdx.y + (blockIdx.y * blockDim.y);
		//posY += StartY;

		Color* mainFrame = (Color*)frame;
		MandelbrotView* mView = (MandelbrotView*)args;
		double Scale = mView->scale;

		int newIndex = (posY * width) + posX;

		Color& target = mainFrame[newIndex];


		int x = posX;
		int y = posY;

		double centerX = -0.5;
		double centerY = 0.5;

		double ResultX;
		double ResultY;

		double Zx = 0;
		double Zy = 0;
		double Zx_x2 = 0;
		double Zy_x2 = 0;

		const double MinimumResultX = centerX - Scale;
		const double MaximumResultX = centerX + Scale;
		const double MinimumResultY = centerY - Scale;
		const double MaximumResultY = centerY + Scale;

		
		double PixelWidth = (MaximumResultX - MinimumResultX) / width;
		double PixelHeight = (MaximumResultY - MinimumResultY) / height;


		int Iteration = 0;
		int MaxIteration = mView->iteration;
		
		const double EscapeRadius = 2.0L;
		double EscapeRadius_x2 = EscapeRadius * EscapeRadius;

		ResultX = (MinimumResultX + PixelWidth * x) + mView->x;
		ResultY = (MinimumResultY + PixelHeight * y) + mView->y;

		if (fabs(ResultY) < PixelHeight / 2) ResultY = 0.0;

		for (; Iteration < MaxIteration && ((Zx_x2 + Zy_x2) < EscapeRadius_x2); Iteration++)
		{
			Zy = 2 * Zx * Zy + ResultY;
			Zx = Zx_x2 - Zy_x2 + ResultX;
			Zx_x2 = Zx * Zx;
			Zy_x2 = Zy * Zy;
		}

		double Value = (1.0 / (double)80) *  Iteration;
		//Ðàíüøå âîçâðàùàëè öâåò ïî òàáëèöå, íî óâû, ýòî íå ýôôåêòèâíûé ñïîñîá
		//return ResoulveColor(Iteration);
		int grayscaleComp = Lerp(0, 255, Value);

		target.R = grayscaleComp; target.G = grayscaleComp; target.B = grayscaleComp;
		target.A = 255;
	}



}