#include "hip/hip_runtime.h"

#define CENGINE_STATUS_INIT -1
#define CENGINE_STATUS_NOCUDA -2
#define CENGINE_STATUS_IDLE 1
#define CENGINE_STATUS_RENDERING 2
#define CENGINE_STATUS_RECONFIGURATE 3
#define CENGINE_STATUS_RENDERFINISHED 4
#define CENGINE_STATUS_FATALERROR 999

//��� ��� �������� (��� ��� �����, ���)
#define CENGINE_MAX_CUDA_RESOURCES 256


extern "C"
{
	//��� ��� � ����� � �������� �������, �?...
	int status;
	int ResourceCounter;
	wchar_t* DebugMessage;

	//device specific
	int cores;

	/*
	* �� ������� ���������� ������ ����� ����������� ��������� ������������� ����� ������� �� ����
	* ��� ������ ��� ���� ��� ������ �������� �� ����: ���� ����� ����� = ���� ����
	* ��� ��������� �� ��������� ������� ����� � ������ �� ���� �����.
	*/
	int height;

	inline int GetBestDeviceID(int Count)
	{
		int curDeviceID = 0;
		int BestComputePower = 0;
		int BestComputeDevice;
		int deviceProhibitenCounter = 0;
		hipDeviceProp_t curDeviceProp;
		while (curDeviceID < Count)
		{
			hipGetDeviceProperties(&curDeviceProp, curDeviceID);
			if (curDeviceProp.computeMode != hipComputeMode::hipComputeModeProhibited)
			{
				if (curDeviceProp.major > 0 && curDeviceProp.major < 9999)
				{
					if (BestComputePower < curDeviceProp.major) BestComputeDevice = curDeviceID;
				}
			}
			else
			{
				deviceProhibitenCounter++;
			}
			curDeviceID++;
		}

		if (deviceProhibitenCounter == Count)
		{
			//������� ������
			DebugMessage = L"All devices in the system prohibiten computeMode. Please google it, and try again.";
			status = CENGINE_STATUS_FATALERROR;
			return -1;
		}
		return BestComputeDevice;
	}
	
	static struct cudaImage
	{
		int width;
		int height;
		size_t bytes;
		void* cudaData;
	};

	bool cuda_init()
	{
		status = CENGINE_STATUS_INIT;
		DebugMessage = L" ";
		hipError_t errcode;
		ResourceCounter = 0;
		//Determine if we have CUDA device?
		int Count;
		hipDeviceProp_t deviceProp;
		errcode = hipGetDeviceCount(&Count);
		if (errcode == hipErrorInsufficientDriver || errcode == hipErrorNoDevice)
		{
			if (errcode == hipErrorInsufficientDriver)
			{
				DebugMessage = L"Cuda driver is outdated! Please update you graphic card driver and try again!";
			}
			else
			{
				DebugMessage = L"There is no cuda device in this machine!";
			}

			status = CENGINE_STATUS_NOCUDA;
			return false;
		}
		int DeviceID;
		errcode = hipGetDevice(&DeviceID);

		//Several devices?
		if (Count > 1)
		{
			DebugMessage = L"Several devices detected! At this point there is no sync code to provide multidevice rendering, sorry.\nHowewer we choose the most powerfull device in the system...";
			//Use a best device. Also check compute_mode prohibiten
			int BestDeviceID = GetBestDeviceID(Count);
			if (BestDeviceID == -1) return false;
		}
		else
		{
			//if only one device, we at least must check is compute mode is prohibiten?
			hipGetDeviceProperties(&deviceProp, DeviceID);
			if (deviceProp.computeMode == hipComputeMode::hipComputeModeProhibited)
			{
				status = CENGINE_STATUS_FATALERROR;
				DebugMessage = L"Device computeMode set to prohibiten! Can't compute!";
				return false;
			}
		}

		//Get device stuff
		

		status = CENGINE_STATUS_IDLE;
		return true;
	}

	cudaImage cuda_AllocTexture(int width, int height)
	{
		cudaImage result;
		result.width = width;
		result.height = height;
		result.bytes = (width * height) * 3;

		ResourceCounter++;
		if (ResourceCounter == CENGINE_MAX_CUDA_RESOURCES)
		{
			status = CENGINE_STATUS_FATALERROR;
			return result;
		}
		
		status = CENGINE_STATUS_IDLE;
		return result;
	}

	void cuda_FreeTexture(cudaImage image)
	{

		ResourceCounter--;
		return;
	}

	void cuda_deinit()
	{
		if (status != CENGINE_STATUS_IDLE)
		{
			//TODO Sync code
		}
		if (ResourceCounter != 0)
		{
			status = CENGINE_STATUS_FATALERROR;
		}
	}

	void cuda_execCode(char* code)
	{

	}



}