#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include "math_functions.h"
#include "EngineCuda_C.cuh"

extern "C"
{
	//Как тут в вашем С устроить объекты, а?...
	int status;
	int ResourceCounter;
	wchar_t* DebugMessage;
	int BestDeviceID;
	hipError_t errcode;
	//device specific
	int cores;

	void* argsMem;

	/*
	* По текущей реализации ширина кадра формируется значением максимального числа потоков на блок
	* Все потому что пока что задача кидается по типу: Одна линия кадра = Один блок
	* Что позволяет по максимуму создать блоки и потоки за один вызов.
	*/
	int height;

	__device__ int getGlobalIdx_1D_2D()
	{
			return blockIdx.x * blockDim.x * blockDim.y
				+ threadIdx.y * blockDim.x + threadIdx.x;
	}

	__device__ int getGlobalIdx_2D_2D()
	{
			int blockId = blockIdx.x
				+ blockIdx.y * gridDim.x;
			int threadId = blockId * (blockDim.x * blockDim.y)
				+ (threadIdx.y * blockDim.x)
				+ threadIdx.x;
			return threadId;
		}

	inline int GetBestDeviceID(int Count)
	{
		int curDeviceID = 0;
		int BestComputePower = 0;
		int BestComputeDevice;
		int deviceProhibitenCounter = 0;
		hipDeviceProp_t curDeviceProp;
		while (curDeviceID < Count)
		{
			hipGetDeviceProperties(&curDeviceProp, curDeviceID);
			if (curDeviceProp.computeMode != hipComputeMode::hipComputeModeProhibited)
			{
				if (curDeviceProp.major > 0 && curDeviceProp.major < 9999)
				{
					if (BestComputePower < curDeviceProp.major) BestComputeDevice = curDeviceID;
				}
			}
			else
			{
				deviceProhibitenCounter++;
			}
			curDeviceID++;
		}

		if (deviceProhibitenCounter == Count)
		{
			//Òÿæåëûé ñëó÷àé
			DebugMessage = L"All devices in the system prohibiten computeMode. Please google it, and try again.";
			status = CENGINE_STATUS_FATALERROR;
			return -1;
		}
		return BestComputeDevice;
	}
	
	const char* GetErrorString(hipError_t errcode)
	{
		return hipGetErrorString(errcode);
	}


	bool cuda_init()
	{
		status = CENGINE_STATUS_INIT;
		DebugMessage = L" ";
		ResourceCounter = 0;
		argsMem = nullptr;
		//Determine if we have CUDA device?
		int Count;
		hipDeviceProp_t deviceProp;
		errcode = hipGetDeviceCount(&Count);
		if (errcode == hipErrorInsufficientDriver || errcode == hipErrorNoDevice)
		{
			if (errcode == hipErrorInsufficientDriver)
			{
				DebugMessage = L"Cuda driver is outdated! Please update you graphic card driver and try again!";
			}
			else
			{
				DebugMessage = L"There is no cuda device in this machine!";
			}

			status = CENGINE_STATUS_NOCUDA;
			return false;
		}
		int DeviceID = 0;

		//Several devices?
		if (Count > 1)
		{
			DebugMessage = L"Several devices detected! At this point there is no sync code to provide multidevice rendering, sorry.\nHowewer we choose the most powerfull device in the system...";
			//Use a best device. Also check compute_mode prohibiten
			BestDeviceID = GetBestDeviceID(Count);
			errcode = hipSetDevice(BestDeviceID);
			if (BestDeviceID == -1) return false;
		}
		else
		{
			//if only one device, we at least must check is compute mode is prohibiten?
			errcode = hipGetDeviceProperties(&deviceProp, DeviceID);
			if (deviceProp.computeMode == hipComputeMode::hipComputeModeProhibited)
			{
				status = CENGINE_STATUS_FATALERROR;
				DebugMessage = L"Device computeMode set to prohibiten! Can't compute!";
				return false;
			}
			BestDeviceID = DeviceID;
			errcode = hipSetDevice(BestDeviceID);
		}

		//Get device stuff
		

		status = CENGINE_STATUS_IDLE;
		return true;
	}



	cudaImage cuda_AllocTexture(int width, int height)
	{
		cudaImage result;
		result.width = width;
		result.height = height;
		result.bytes = (width * height) * 3;
		
		ResourceCounter++;
		if (ResourceCounter == CENGINE_MAX_CUDA_RESOURCES)
		{
			status = CENGINE_STATUS_FATALERROR;
			return result;
		}
		
		status = CENGINE_STATUS_IDLE;
		return result;
	}

	void cuda_FreeTexture(cudaImage image)
	{

		ResourceCounter--;
		return;
	}

	void cuda_deinit()
	{
		if (status != CENGINE_STATUS_IDLE)
		{
			//TODO Sync code
		}
		if (ResourceCounter != 0)
		{
			status = CENGINE_STATUS_FATALERROR;
		}
	}

	void cuda_execCode(char* code)
	{

	}
	void WaitCudaThread()
	{
		hipDeviceSynchronize();
	}

	hipError_t temp_callKernels(int width, int height, pFrame frame, void* args, int argsSize)
	{
		dim3 grid (256, 256, 1);
		dim3 blocks(width / grid.x, height / grid.y, 1);
		if (argsMem != nullptr) hipFree(argsMem);
		
		errcode = hipConfigureCall(grid, blocks);

		errcode = hipSetupArgument(&width, sizeof(int), 0);
		errcode = hipSetupArgument(&height,sizeof(int), sizeof (int));
		errcode = hipSetupArgument(&frame,sizeof(pFrame), sizeof (int)* 2);
		if (args != nullptr)
		{
			errcode = hipMalloc(&argsMem, argsSize);
			errcode = hipMemcpy(argsMem, args, argsSize, hipMemcpyKind::hipMemcpyHostToDevice);
			errcode = hipSetupArgument(&argsMem, sizeof(void*), sizeof(int) * 2 + sizeof(pFrame));
		}


		errcode = hipLaunchByPtr(testKernelFunc);

		return errcode;
		//testKernelFunc <<<grid, blocks >>> (width, height, frame);
	}

	__device__ inline int Lerp(int start, int end, double value)
	{
		return start + (end - start) * value;
	}

	//dim3 blockIdx <- gridDim
	//dim3 threadIdx <- blocks

	__global__ void testKernelFunc(int width, int height, pFrame frame, void* args)
	{
		//init pointers and pixel coord
		//compute target pixel index for thread and block
		int posX = threadIdx.x + (blockIdx.x * blockDim.x);
		int posY = threadIdx.y + (blockIdx.y * blockDim.y);

		Color* mainFrame = (Color*)frame;
		MandelbrotView* mView = (MandelbrotView*)args;
		double Scale = mView->scale;

		int newIndex = (posY * width) + posX;

		Color& target = mainFrame[newIndex];


		int x = posX;
		int y = posY;

		double centerX = -0.5;
		double centerY = 0.5;

		double ResultX;
		double ResultY;

		double Zx = 0;
		double Zy = 0;
		double Zx_x2 = 0;
		double Zy_x2 = 0;

		const double MinimumResultX = centerX - Scale;
		const double MaximumResultX = centerX + Scale;
		const double MinimumResultY = centerY - Scale;
		const double MaximumResultY = centerY + Scale;

		
		double PixelWidth = (MaximumResultX - MinimumResultX) / width;
		double PixelHeight = (MaximumResultY - MinimumResultY) / height;


		int Iteration = 0;
		int MaxIteration = mView->iteration;
		
		const double EscapeRadius = 2.0L;
		double EscapeRadius_x2 = EscapeRadius * EscapeRadius;

		ResultX = (MinimumResultX + PixelWidth * x) + mView->x;
		ResultY = (MinimumResultY + PixelHeight * y) + mView->y;

		if (fabs(ResultY) < PixelHeight / 2) ResultY = 0.0;

		for (; Iteration < MaxIteration && ((Zx_x2 + Zy_x2) < EscapeRadius_x2); Iteration++)
		{
			Zy = 2 * Zx * Zy + ResultY;
			Zx = Zx_x2 - Zy_x2 + ResultX;
			Zx_x2 = Zx * Zx;
			Zy_x2 = Zy * Zy;
		}

		double Value = (1.0 / (double)80) *  Iteration;
		//Ðàíüøå âîçâðàùàëè öâåò ïî òàáëèöå, íî óâû, ýòî íå ýôôåêòèâíûé ñïîñîá
		//return ResoulveColor(Iteration);
		int grayscaleComp = Lerp(0, 255, Value);

		target.R = grayscaleComp; target.G = grayscaleComp; target.B = grayscaleComp;
		target.A = 255;
	}



}